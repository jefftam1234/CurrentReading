#include "hip/hip_runtime.h"
//
// Created by jeff on 1/13/23.
//

#include "MatrixDecomposition.cuh"
#include "CudaMarcos.cuh"
#include "CudaUtil.h"
#include <hipblas.h>
#include <hipsolver.h>
#include <vector>
#include "VectorUtil.cuh"

using namespace std;


vector<vector<vector<double>>> JTQuant::Math::householderQR_CPU(vector<vector<double>> A) {
    int m = A.size();
    int n = A[0].size();
    vector<vector<double> > A_mod = A;
    vector<vector<double> > Q = Math::identity(m);

    for (int k = 0; k < n; k++) {
        //printf("k = %d\n", k);
        int e_size = m - k;
        //copy the vector x from the original matrix
        vector<double> x(e_size);
        for (int i = k; i < m; i++) x[i - k] = A_mod[i][k];
        vector<double> e(e_size, 0);
        e[0] = 1;
        double x_norm = sqrt(Math::inner(x, x));
        double sign_x = x[0] >= 0 ? 1 : -1;
        // create householder reflector vector u
        // (function) vector<double> u = sum(product(sign_x * x_norm, e), x);
        vector<double> u(e_size);
        for (int i = 0; i < e_size; i++) u[i] = sign_x * x_norm * e[i] + x[i];
        // v is the normalized vector u
        double v_norm = sqrt(Math::inner(u, u));
        vector<double> v(e_size, 0);
        for (int i = 0; i < e_size; i++) v[i] = u[i] / v_norm;

        vector<vector<double> > H(e_size, vector<double>(e_size, 0));
        for (int i = 0; i < e_size; i++) H[i][i] = 1;
        for (int i = 0; i < e_size; i++)
            for (int j = 0; j < e_size; j++)
                H[i][j] -= 2 * v[i] * v[j];

        vector<vector<double>> temp(e_size, vector<double>(n - k, 0));
        for (int i = 0; i < e_size; i++) {
            for (int j = 0; j < n - k; j++) {
                for (int l = 0; l < e_size; l++) {
                    //temp[i][j] += H[i][l] * A_mod[k + l][k + j];
                    double H_value = H[i][l];
                    double A_value = A_mod[k + l][k + j];
                    double res = H_value * A_value;
                    temp[i][j] += res;
//                    cout << "temp[" << i << "][" << j << "] += H[" << i << "][" << l << "] * A_mod[" << k + l << "]["
//                         << k + j
//                         << "] = " << H_value << " * " << A_value << " = " << res << endl;
                }
                //cout << "temp[" << i << "][" << j << "] = " << temp[i][j] << endl;
            }
        }


        for (int i = 0; i < e_size; i++)
            for (int j = 0; j < n - k; j++)
                A_mod[k + i][k + j] = temp[i][j];

        //printMatrixCPU(temp, "temp");

        // update Q with the current householder matrix
        vector<vector<double> > H_p(m, vector<double>(m, 0));
        for (int i = 0; i < m; i++) H_p[i][i] = 1;
        for (int i = 0; i < e_size; i++)
            for (int j = 0; j < e_size; j++)
                H_p[m - e_size + i][m - e_size + j] = H[i][j];

        vector<vector<double> > Q_temp(m, vector<double>(m, 0));
        for (int i = 0; i < m; i++) {
            for (int j = 0; j < m; j++) {
                for (int l = 0; l < m; l++) {
                    Q_temp[i][j] += Q[i][l] * H_p[l][j];
                }
            }
        }
        Q = Q_temp;
    }

    vector<vector<double>> R(n, vector<double>(n, 0));
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++)
            R[i][j] = A_mod[i][j];
    return {Q, R};
}

__device__ void
JTQuant::Math::householderQR_device(double *A, int m, int n, double *x, double *e, double *u, double *v, double *H,
                                    double *temp) {
    for (int k = 0; k < n; k++) {
        //printf("k = %d\n", k);
        int e_size = m - k;
        for (int i = 0; i < e_size; i++) {
            if (i == 0) {
                e[i] = 1.0;
                continue;
            }
            e[i] = 0.0;
        }
        for (int i = k; i < m; i++) x[i - k] = A[i * n + k];
        double x_norm = 0;
        for (int i = 0; i < e_size; i++) x_norm += x[i] * x[i];
        x_norm = sqrt(x_norm);
        double sign_x = 0.0;

        if (x[0] >= 0) sign_x = 1;
        else sign_x = -1;

        for (int i = 0; i < e_size; i++) u[i] = sign_x * x_norm * e[i] + x[i];
        double u_norm = 0;
        for (int i = 0; i < e_size; i++) u_norm += u[i] * u[i];
        u_norm = sqrt(u_norm);
        for (int i = 0; i < e_size; i++) v[i] = u[i] / u_norm;


        //printVectorGPU(v, e_size, "v");

        for (int i = 0; i < e_size; i++) {
            for (int j = 0; j < e_size; j++) {
                H[i * e_size + j] = (i == j) ? 1 : 0;
                H[i * e_size + j] -= 2 * v[i] * v[j];
            }
        }

        //printMatrixGPU(H, e_size, e_size, "H");

        for (int i = 0; i < e_size; i++) {
            for (int j = 0; j < n - k; j++) {
                for (int l = 0; l < e_size; l++) {
                    //temp[i * (n - k) + j] += H[i * e_size + l] * A[(k + l) * n + (k + j)];   //problem here, step 2, H is right, A should be right (indicing is wrong?)
                    double H_value = H[i * e_size + l];
                    double A_value = A[(k + l) * n + (k + j)];
                    double res = H_value * A_value;
                    temp[i * (n - k) + j] += res;
                    //printf("temp[%d][%d] += H[%d][%d] * A[%d][%d] = %f * %f = %f\n", i, j, i, l, k + l, k + j, H_value, A_value, res);
                }
                //printf("temp[%d][%d] = %f\n", i, j, temp[i * (n - k) + j]);
            }
        }

        for (int i = 0; i < e_size; i++) {
            for (int j = 0; j < n - k; j++) {
                A[(k + i) * n + (k + j)] = temp[i * (n - k) + j];
                temp[i * (n - k) + j] = 0.0;   //initialized
            }
        }
        //printMatrixGPU(temp, e_size, n - k, "temp");
    }
}

__global__ void
JTQuant::Math::householderQR_global(double *A, int m, int n, double *x, double *e, double *u, double *v, double *H,
                                    double *temp) {
    householderQR_device(A, m, n, x, e, u, v, H, temp);
}

