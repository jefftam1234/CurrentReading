#include "hip/hip_runtime.h"
//
// Created by jeff on 1/13/23.
//


#include <iostream>
#include <iomanip>
#include "MatrixDecompositionTest.cuh"

//sum of two vectors
vector<double> JTQuant::TestSuites::sum(vector<double> a, vector<double> b) {
    int n = a.size();
    vector<double> c(n);
    for (int i = 0; i < n; i++) c[i] = a[i] + b[i];
    return c;
}

//minus of two vectors
vector<double> JTQuant::TestSuites::minus(vector<double> a, vector<double> b) {
    int n = a.size();
    vector<double> c(n);
    for (int i = 0; i < n; i++) c[i] = a[i] - b[i];
    return c;
}

//inner product of two vectors
double JTQuant::TestSuites::inner(vector<double> a, vector<double> b) {
    int n = a.size();
    double res = 0;
    for (int i = 0; i < n; i++) res += a[i] * b[i];
    return res;
}

//outer product of two vectors
vector<vector<double>> JTQuant::TestSuites::outer(vector<double> a, vector<double> b) {
    int n = a.size();
    int m = b.size();
    vector<vector<double> > res(n, vector<double>(m, 0));
    for (int i = 0; i < n; i++)
        for (int j = 0; j < m; j++)
            res[i][j] = a[i] * b[j];
    return res;
}

vector<vector<double>> JTQuant::TestSuites::transpose(vector<vector<double> > A) {
    int m = A.size();
    int n = A[0].size();
    vector<vector<double> > AT(n, vector<double>(m, 0));
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            AT[j][i] = A[i][j];
        }
    }
    return AT;
}

//scalar product with a vector
vector<double> JTQuant::TestSuites::product(double c, vector<double> b) {
    int n = b.size();
    vector<double> res(n, 0);
    for (int i = 0; i < n; i++) res[i] = c * b[i];
    return res;
}

//matrix product with a vector
vector<double> JTQuant::TestSuites::product(vector<vector<double> > M, vector<double> b) {
    int n = M.size();
    int m = b.size();
    vector<double> res(n, 0);
    for (int i = 0; i < n; i++)
        for (int j = 0; j < m; j++)
            res[i] += M[i][j] * b[j];
    return res;
}

vector<vector<double>> JTQuant::TestSuites::product(vector<vector<double>> M, vector<vector<double>> N) {
    // function to return the product of two matrices
    int m = M.size();
    int n = N[0].size();
    int p = N.size();
    vector<vector<double> > C(m, vector<double>(n, 0));
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
            for (int k = 0; k < p; k++)
                C[i][j] += M[i][k] * N[k][j];
    return C;
}


vector<vector<double>> JTQuant::TestSuites::identity(int n) {
    vector<vector<double>> I(n, vector<double>(n, 0));
    for (int i = 0; i < n; i++)
        I[i][i] = 1;
    return I;
}

int JTQuant::TestSuites::vector_util_test() {
    cout << "testing vector_util_test.\n";
    vector<double> a = {1, 2, 3};
    vector<double> b = {4, 5, 6};
    cout << "vector a: ";
    for (int i = 0; i < a.size(); i++) cout << a[i] << " ";
    cout << endl;
    cout << "vector b: ";
    for (int i = 0; i < b.size(); i++) cout << b[i] << " ";
    cout << endl;

    vector<double> c = sum(a, b);
    cout << "Sum of a and b: ";
    for (int i = 0; i < c.size(); i++) cout << c[i] << " ";
    cout << endl;

    vector<double> d = minus(a, b);
    cout << "Difference of a and b: ";
    for (int i = 0; i < d.size(); i++) cout << d[i] << " ";
    cout << endl;

    double e = inner(a, b);
    cout << "Inner product of a and b: " << e << endl;

    vector<vector<double> > f = outer(a, b);
    cout << "Outer product of a and b: " << endl;
    for (int i = 0; i < f.size(); i++) {
        for (int j = 0; j < f[0].size(); j++) cout << f[i][j] << " ";
        cout << endl;
    }

    vector<double> g = product(f, c);
    cout << "Product of f and c: ";
    for (int i = 0; i < g.size(); i++) cout << g[i] << " ";
    cout << endl;

    vector<vector<double> > A = {{1, 2, 3},
                                 {4, 5, 6}};
    vector<vector<double> > B = {{1, 2},
                                 {3, 4},
                                 {5, 6}};

    vector<vector<double> > C = product(A, B);
    cout << "Product of A and B:" << endl;
    for (int i = 0; i < C.size(); i++) {
        for (int j = 0; j < C[0].size(); j++) cout << C[i][j] << " ";
        cout << endl;
    }

    return 0;
}

vector<vector<vector<double>>> JTQuant::TestSuites::householderQR2(vector<vector<double>> A) {
    int m = A.size();
    int n = A[0].size();
    vector<vector<double> > A_mod = A;
    vector<vector<double> > Q = identity(m);

    for (int k = 0; k < n; k++) {
        int e_size = m - k;
        //copy the vector x from the original matrix
        vector<double> x(e_size);
        for (int i = k; i < m; i++) x[i - k] = A_mod[i][k];
        vector<double> e(e_size, 0);
        e[0] = 1;
        double x_norm = sqrt(inner(x, x));
        double sign_x = x[0] >= 0 ? 1 : -1;
        // create householder reflector vector u
        // (function) vector<double> u = sum(product(sign_x * x_norm, e), x);
        vector<double> u(e_size);
        for (int i = 0; i < e_size; i++) u[i] = sign_x * x_norm * e[i] + x[i];
        // v is the normalized vector u
        double v_norm = sqrt(inner(u, u));
        vector<double> v(e_size, 0);
        for (int i = 0; i < e_size; i++) v[i] = u[i] / v_norm;

        vector<vector<double> > H(e_size, vector<double>(e_size, 0));
        for (int i = 0; i < e_size; i++) H[i][i] = 1;
        for (int i = 0; i < e_size; i++)
            for (int j = 0; j < e_size; j++)
                H[i][j] -= 2 * v[i] * v[j];

        vector<vector<double> > temp(e_size, vector<double>(n - k, 0));
        for (int i = 0; i < e_size; i++)
            for (int j = 0; j < n - k; j++)
                for (int l = 0; l < e_size; l++)
                    temp[i][j] += H[i][l] * A_mod[k + l][k + j];

        for (int i = 0; i < e_size; i++)
            for (int j = 0; j < n - k; j++)
                A_mod[k + i][k + j] = temp[i][j];

        // update Q with the current householder matrix
        vector<vector<double> > H_p(m, vector<double>(m, 0));
        for (int i = 0; i < m; i++) H_p[i][i] = 1;
        for (int i = 0; i < e_size; i++)
            for (int j = 0; j < e_size; j++)
                H_p[m - e_size + i][m - e_size + j] = H[i][j];

        vector<vector<double> > Q_temp(m, vector<double>(m, 0));
        for (int i = 0; i < m; i++) {
            for (int j = 0; j < m; j++) {
                for (int l = 0; l < m; l++) {
                    Q_temp[i][j] += Q[i][l] * H_p[l][j];
                }
            }
        }
        Q = Q_temp;
    }

    vector<vector<double>> R(n, vector<double>(n, 0));
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++)
            R[i][j] = A_mod[i][j];
    return {Q, R};
}

int JTQuant::TestSuites::householder_qr_test() {
    vector<vector<double> > A = {
            {1.0, -1.0, 4.0},
            {1.0, 4.0,  -2.0},
            {1.0, 4.0,  2.0},
            {1.0, -1.0, 0.0}
    };

    vector<vector<vector<double>>> QR = householderQR2(A);
    vector<vector<double>> Q = QR[0];
    vector<vector<double>> R = QR[1];

    cout << "Q: " << endl;
    for (int i = 0; i < Q.size(); i++) {
        for (int j = 0; j < Q[0].size(); j++) {
            cout << setw(12) << setfill(' ') << fixed << setprecision(6) << Q[i][j];
        }
        cout << endl;
    }

    cout << "R:" << endl;
    for (int i = 0; i < R.size(); i++) {
        for (int j = 0; j < R[0].size(); j++)
            cout << setw(12) << setfill(' ') << fixed << setprecision(6) << R[i][j];
        cout << endl;
    }

    vector<vector<double>> transpose_R = transpose(R);

    vector<vector<double>> RTR = product(transpose_R, R);

    cout << "R:" << endl;
    for (int i = 0; i < RTR.size(); i++) {
        for (int j = 0; j < RTR[0].size(); j++)
            cout << setw(12) << setfill(' ') << fixed << setprecision(6) << RTR[i][j];
        cout << endl;
    }

    return 0;
}




