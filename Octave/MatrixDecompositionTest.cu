#include "hip/hip_runtime.h"
//
// Created by jeff on 1/13/23.
//


#include <iostream>
#include <iomanip>
#include <cassert>
#include "MatrixDecompositionTest.cuh"
#include "VectorUtil.cuh"
#include "MatrixDecomposition.cuh"

#define EPSILON 1e-8

using namespace std;

int JTQuant::TestSuites::vector_util_test() {
    vector<double> a = {1, 2, 3};
    vector<double> b = {4, 5, 6};
    vector<double> c = Math::sum(a, b);
    assert(c[0] == 5 && c[1] == 7 && c[2] == 9);
    vector<double> d = Math::minus(a, b);
    assert(d[0] == -3 && d[1] == -3 && d[2] == -3);
    double e = Math::inner(a, b);
    assert(e == 32);
    vector<vector<double> > f = Math::outer(a, b);
    assert(f[0][0] == 4 && f[0][1] == 5 && f[0][2] == 6);
    assert(f[1][0] == 8 && f[1][1] == 10 && f[1][2] == 12);
    assert(f[2][0] == 12 && f[2][1] == 15 && f[2][2] == 18);
    vector<double> g = Math::product(f, c);
    assert(g[0] == 109 && g[1] == 218 && g[2] == 327);
    vector<vector<double> > A = {{1, 2, 3},
                                 {4, 5, 6}};
    vector<vector<double> > B = {{1, 2},
                                 {3, 4},
                                 {5, 6}};

    vector<vector<double> > C = Math::product(A, B);
    assert(C[0][0] == 22 && C[0][1] == 28);
    assert(C[1][0] == 49 && C[1][1] == 64);

    return 0;
}


int JTQuant::TestSuites::householder_qr_cpu_test() {
    vector<vector<double> > A = {
            {1.0, -1.0, 4.0},
            {1.0, 4.0,  -2.0},
            {1.0, 4.0,  2.0},
            {1.0, -1.0, 0.0}
    };

    vector<vector<vector<double>>> QR = JTQuant::Math::householderQR_CPU(A);
    vector<vector<double>> Q = QR[0];
    vector<vector<double>> R = QR[1];

//    cout << "Q: " << endl;
//    for (int i = 0; i < Q.size(); i++) {
//        for (int j = 0; j < Q[0].size(); j++) {
//            cout << setw(12) << setfill(' ') << fixed << setprecision(6) << Q[i][j];
//        }
//        cout << endl;
//    }
//
//    cout << "R:" << endl;
//    for (int i = 0; i < R.size(); i++) {
//        for (int j = 0; j < R[0].size(); j++)
//            cout << setw(12) << setfill(' ') << fixed << setprecision(6) << R[i][j];
//        cout << endl;
//    }

    vector<vector<double>> transpose_R = Math::transpose(R);
    vector<vector<double>> RTR = Math::product(transpose_R, R);

    vector<vector<double>> expected_R = {
            {-2.0, -3.0, -2.0},
            {0.0,  -5.0, 2.0},
            {0.0,  0.0,  -4.0}
    };
// Assert statement
    for (int i = 0; i < R.size(); i++) {
        for (int j = 0; j < R[0].size(); j++) assert(fabs(R[i][j] - expected_R[i][j]) < EPSILON);
    }

//    cout << "RTR:" << endl;
//    for (int i = 0; i < RTR.size(); i++) {
//        for (int j = 0; j < RTR[0].size(); j++)
//            cout << setw(12) << setfill(' ') << fixed << setprecision(6) << RTR[i][j];
//        cout << endl;
//    }

    return 0;
}


int JTQuant::TestSuites::householder_qr_gpu_test() {

    vector<vector<double>> A_h = {
            {1.0, -1.0, 4.0},
            {1.0, 4.0,  -2.0},
            {1.0, 4.0,  2.0},
            {1.0, -1.0, 0.0}
    };
    int m = A_h.size(); // number of rows in the matrix A
    int n = A_h[0].size(); // number of columns in the matrix A

    double *A_d; // device array for the matrix A
    double *x, *e, *u, *v, *H, *temp;
    hipMallocManaged((void **) &A_d, m * n * sizeof(double));
    hipMallocManaged((void **) &x, m * sizeof(double));
    hipMallocManaged((void **) &e, m * sizeof(double));
    hipMallocManaged((void **) &u, m * sizeof(double));
    hipMallocManaged((void **) &v, m * sizeof(double));
    hipMallocManaged((void **) &H, m * m * sizeof(double));
    hipMallocManaged((void **) &temp, m * (n - 0) * sizeof(double));
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            A_d[i * n + j] = A_h[i][j];
        }
    }
    dim3 grid(1, 1);
    dim3 block(1, 1);
    JTQuant::Math::householderQR_global<<<grid, block>>>(A_d, m, n, x, e, u, v, H, temp);
    hipDeviceSynchronize();

//    cout << "R:" << endl;
//    for (int i = 0; i < m; i++) {
//        for (int j = 0; j < n; j++)
//            cout << setw(12) << setfill(' ') << fixed << setprecision(6) << A_d[i * n + j];
//        cout << endl;
//    }
    hipFree(x);
    hipFree(e);
    hipFree(u);
    hipFree(v);
    hipFree(H);
    hipFree(temp);
    // Parse matrix
    vector<vector<double>> expected_R = {
            {-2.0, -3.0, -2.0},
            {0.0,  -5.0, 2.0},
            {0.0,  0.0,  -4.0},
            {0.0,  0.0,  0.0}
    };
    // Assert statement
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            double expected = expected_R[i][j];
            assert(fabs(A_d[i * n + j] - expected) < EPSILON);
        }
    }
    return 0;
}




