#include "hip/hip_runtime.h"
//
// Created by jeff on 1/13/23.
//


#include <iostream>
#include <iomanip>
#include "MatrixDecompositionTest.cuh"

//sum of two vectors
vector<double> JTQuant::TestSuites::sum(vector<double> a, vector<double> b) {
    int n = a.size();
    vector<double> c(n);
    for (int i = 0; i < n; i++) c[i] = a[i] + b[i];
    return c;
}

//minus of two vectors
vector<double> JTQuant::TestSuites::minus(vector<double> a, vector<double> b) {
    int n = a.size();
    vector<double> c(n);
    for (int i = 0; i < n; i++) c[i] = a[i] - b[i];
    return c;
}

//inner product of two vectors
double JTQuant::TestSuites::inner(vector<double> a, vector<double> b) {
    int n = a.size();
    double res = 0;
    for (int i = 0; i < n; i++) res += a[i] * b[i];
    return res;
}

//outer product of two vectors
vector<vector<double>> JTQuant::TestSuites::outer(vector<double> a, vector<double> b) {
    int n = a.size();
    int m = b.size();
    vector<vector<double> > res(n, vector<double>(m, 0));
    for (int i = 0; i < n; i++)
        for (int j = 0; j < m; j++)
            res[i][j] = a[i] * b[j];
    return res;
}

vector<vector<double>> JTQuant::TestSuites::transpose(vector<vector<double> > A) {
    int m = A.size();
    int n = A[0].size();
    vector<vector<double> > AT(n, vector<double>(m, 0));
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            AT[j][i] = A[i][j];
        }
    }
    return AT;
}

//scalar product with a vector
vector<double> JTQuant::TestSuites::product(double c, vector<double> b) {
    int n = b.size();
    vector<double> res(n, 0);
    for (int i = 0; i < n; i++) res[i] = c * b[i];
    return res;
}

//matrix product with a vector
vector<double> JTQuant::TestSuites::product(vector<vector<double> > M, vector<double> b) {
    int n = M.size();
    int m = b.size();
    vector<double> res(n, 0);
    for (int i = 0; i < n; i++)
        for (int j = 0; j < m; j++)
            res[i] += M[i][j] * b[j];
    return res;
}

vector<vector<double>> JTQuant::TestSuites::product(vector<vector<double>> M, vector<vector<double>> N) {
    // function to return the product of two matrices
    int m = M.size();
    int n = N[0].size();
    int p = N.size();
    vector<vector<double> > C(m, vector<double>(n, 0));
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
            for (int k = 0; k < p; k++)
                C[i][j] += M[i][k] * N[k][j];
    return C;
}


vector<vector<double>> JTQuant::TestSuites::identity(int n) {
    vector<vector<double>> I(n, vector<double>(n, 0));
    for (int i = 0; i < n; i++)
        I[i][i] = 1;
    return I;
}

int JTQuant::TestSuites::vector_util_test() {
    cout << "testing vector_util_test.\n";
    vector<double> a = {1, 2, 3};
    vector<double> b = {4, 5, 6};
    cout << "vector a: ";
    for (int i = 0; i < a.size(); i++) cout << a[i] << " ";
    cout << endl;
    cout << "vector b: ";
    for (int i = 0; i < b.size(); i++) cout << b[i] << " ";
    cout << endl;

    vector<double> c = sum(a, b);
    cout << "Sum of a and b: ";
    for (int i = 0; i < c.size(); i++) cout << c[i] << " ";
    cout << endl;

    vector<double> d = minus(a, b);
    cout << "Difference of a and b: ";
    for (int i = 0; i < d.size(); i++) cout << d[i] << " ";
    cout << endl;

    double e = inner(a, b);
    cout << "Inner product of a and b: " << e << endl;

    vector<vector<double> > f = outer(a, b);
    cout << "Outer product of a and b: " << endl;
    for (int i = 0; i < f.size(); i++) {
        for (int j = 0; j < f[0].size(); j++) cout << f[i][j] << " ";
        cout << endl;
    }

    vector<double> g = product(f, c);
    cout << "Product of f and c: ";
    for (int i = 0; i < g.size(); i++) cout << g[i] << " ";
    cout << endl;

    vector<vector<double> > A = {{1, 2, 3},
                                 {4, 5, 6}};
    vector<vector<double> > B = {{1, 2},
                                 {3, 4},
                                 {5, 6}};

    vector<vector<double> > C = product(A, B);
    cout << "Product of A and B:" << endl;
    for (int i = 0; i < C.size(); i++) {
        for (int j = 0; j < C[0].size(); j++) cout << C[i][j] << " ";
        cout << endl;
    }

    return 0;
}

vector<vector<vector<double>>> JTQuant::TestSuites::householderQR(vector<vector<double>> A) {
    int m = A.size();
    int n = A[0].size();
    vector<vector<double> > A_mod = A;
    vector<vector<double> > Q = identity(m);

    for (int k = 0; k < n; k++) {
        int e_size = m - k;
        //copy the vector x from the original matrix
        vector<double> x(e_size);
        for (int i = k; i < m; i++) x[i - k] = A_mod[i][k];
        vector<double> e(e_size, 0);
        e[0] = 1;
        double x_norm = sqrt(inner(x, x));
        double sign_x = x[0] >= 0 ? 1 : -1;
        // create householder reflector vector u
        // (function) vector<double> u = sum(product(sign_x * x_norm, e), x);
        vector<double> u(e_size);
        for (int i = 0; i < e_size; i++) u[i] = sign_x * x_norm * e[i] + x[i];
        // v is the normalized vector u
        double v_norm = sqrt(inner(u, u));
        vector<double> v(e_size, 0);
        for (int i = 0; i < e_size; i++) v[i] = u[i] / v_norm;

        vector<vector<double> > H(e_size, vector<double>(e_size, 0));
        for (int i = 0; i < e_size; i++) H[i][i] = 1;
        for (int i = 0; i < e_size; i++)
            for (int j = 0; j < e_size; j++)
                H[i][j] -= 2 * v[i] * v[j];

        vector<vector<double> > temp(e_size, vector<double>(n - k, 0));
        for (int i = 0; i < e_size; i++)
            for (int j = 0; j < n - k; j++)
                for (int l = 0; l < e_size; l++)
                    temp[i][j] += H[i][l] * A_mod[k + l][k + j];

        for (int i = 0; i < e_size; i++)
            for (int j = 0; j < n - k; j++)
                A_mod[k + i][k + j] = temp[i][j];

        // update Q with the current householder matrix
        vector<vector<double> > H_p(m, vector<double>(m, 0));
        for (int i = 0; i < m; i++) H_p[i][i] = 1;
        for (int i = 0; i < e_size; i++)
            for (int j = 0; j < e_size; j++)
                H_p[m - e_size + i][m - e_size + j] = H[i][j];

        vector<vector<double> > Q_temp(m, vector<double>(m, 0));
        for (int i = 0; i < m; i++) {
            for (int j = 0; j < m; j++) {
                for (int l = 0; l < m; l++) {
                    Q_temp[i][j] += Q[i][l] * H_p[l][j];
                }
            }
        }
        Q = Q_temp;
    }

    vector<vector<double>> R(n, vector<double>(n, 0));
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++)
            R[i][j] = A_mod[i][j];
    return {Q, R};
}

int JTQuant::TestSuites::householder_qr_cpu_test() {
    vector<vector<double> > A = {
            {1.0, -1.0, 4.0},
            {1.0, 4.0,  -2.0},
            {1.0, 4.0,  2.0},
            {1.0, -1.0, 0.0}
    };

    vector<vector<vector<double>>> QR = householderQR(A);
    vector<vector<double>> Q = QR[0];
    vector<vector<double>> R = QR[1];

    cout << "Q: " << endl;
    for (int i = 0; i < Q.size(); i++) {
        for (int j = 0; j < Q[0].size(); j++) {
            cout << setw(12) << setfill(' ') << fixed << setprecision(6) << Q[i][j];
        }
        cout << endl;
    }

    cout << "R:" << endl;
    for (int i = 0; i < R.size(); i++) {
        for (int j = 0; j < R[0].size(); j++)
            cout << setw(12) << setfill(' ') << fixed << setprecision(6) << R[i][j];
        cout << endl;
    }

    vector<vector<double>> transpose_R = transpose(R);

    vector<vector<double>> RTR = product(transpose_R, R);

    cout << "RTR:" << endl;
    for (int i = 0; i < RTR.size(); i++) {
        for (int j = 0; j < RTR[0].size(); j++)
            cout << setw(12) << setfill(' ') << fixed << setprecision(6) << RTR[i][j];
        cout << endl;
    }

    return 0;
}

//__device__ void householderQR_kernel(double *A, int m, int n) {
//    for (int k = 0; k < n; k++) {
//        int e_size = m - k;
//        double x[e_size];
//        double e[e_size];
//        double u[e_size];
//        double v[e_size];
//        double H[e_size][e_size];
//        double temp[e_size][n - k];
//        double x_norm, u_norm;
//        for (int i = k; i < m; i++) x[i - k] = A[i * n + k];
//        e[0] = 1;
//        for (int i = 0; i < e_size; i++) x_norm += x[i] * x[i];
//        x_norm = sqrt(x_norm);
//        double sign_x = x[0] >= 0 ? 1 : -1;
//        for (int i = 0; i < e_size; i++) u[i] = sign_x * x_norm * e[i] + x[i];
//        for (int i = 0; i < e_size; i++) u_norm += u[i] * u[i];
//        u_norm = sqrt(u_norm);
//        for (int i = 0; i < e_size; i++) v[i] = u[i] / u_norm;
//
//
//        for (int i = 0; i < e_size; i++) {
//            for (int j = 0; j < e_size; j++) {
//                H[i][j] = (i == j) ? 1 : 0;
//                H[i][j] -= 2 * v[i] * v[j];
//            }
//        }
//
//
//        for (int i = 0; i < e_size; i++) {
//            for (int j = 0; j < n - k; j++) {
//                for (int l = 0; l < e_size; l++) {
//                    temp[i][j] += H[i][l] * A[(k + l) * n + (k + j)];
//                }
//            }
//        }
//
//        for (int i = 0; i < e_size; i++) {
//            for (int j = 0; j < n - k; j++) {
//                A[(k + i) * n + (k + j)] = temp[i][j];
//            }
//        }
//    }
//}
__device__ void print_vector(double *vec, int size, const char *name) {
  printf("%s:\n", name);
  for (int i = 0; i < size; i++) {
    printf("  vec[%d] = %f\n", i, vec[i]);
  }
}

__device__ void print_matrix(double *mat, int rows, int cols, const char *name) {
  printf("%s:\n", name);
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < cols; j++) {
      printf("  mat[%d][%d] = %f\n", i, j, mat[i * cols + j]);
    }
  }
}

__device__ void
householderQR_device(double *A, int m, int n, double *x, double *e, double *u, double *v, double *H, double *temp) {
    for (int k = 0; k < n; k++) {
        printf("k = %d\n", k);
        int e_size = m - k;
        for (int i = 0; i < e_size; i++) {
            if (i == 0) {
                e[i] = 1.0;
                continue;
            }
            e[i] = 0.0;
        }
        for (int i = k; i < m; i++) x[i - k] = A[i * n + k];
        double x_norm = 0;
        for (int i = 0; i < e_size; i++) x_norm += x[i] * x[i];
        x_norm = sqrt(x_norm);
        double sign_x = 0.0;

        if (x[0] >= 0) sign_x = 1;
        else sign_x = -1;

        for (int i = 0; i < e_size; i++) u[i] = sign_x * x_norm * e[i] + x[i];
        double u_norm = 0;
        for (int i = 0; i < e_size; i++) u_norm += u[i] * u[i];
        u_norm = sqrt(u_norm);
        for (int i = 0; i < e_size; i++) v[i] = u[i] / u_norm;


        print_vector(v, e_size, "v");

        for (int i = 0; i < e_size; i++) {
            for (int j = 0; j < e_size; j++) {
                H[i * e_size + j] = (i == j) ? 1 : 0;
                H[i * e_size + j] -= 2 * v[i] * v[j];
            }
        }

        print_matrix(H, e_size, e_size, "H");

        for (int i = 0; i < e_size; i++) {
            for (int j = 0; j < n - k; j++) {
                for (int l = 0; l < e_size; l++) {
                    temp[i * (n - k) + j] += H[i * e_size + l] * A[(k + l) * n + (k + j)];   //problem here, step 2, H is right, A should be right (indicing is wrong?)
                }
            }
        }

        for (int i = 0; i < e_size; i++) {
            for (int j = 0; j < n - k; j++) {
                A[(k + i) * n + (k + j)] = temp[i * (n - k) + j];
            }
        }
        print_matrix(temp, e_size, n - k, "temp");
    }
}

__global__ void
householderQR_global(double *A, int m, int n, double *x, double *e, double *u, double *v, double *H, double *temp) {
    householderQR_device(A, m, n, x, e, u, v, H, temp);
}

int JTQuant::TestSuites::householder_qr_gpu_test() {
    int m = 4; // number of rows in the matrix A
    int n = 3; // number of columns in the matrix A
    vector<vector<double>> A_h = {
            {1.0, -1.0, 4.0},
            {1.0, 4.0,  -2.0},
            {1.0, 4.0,  2.0},
            {1.0, -1.0, 0.0}
    };

    double *A_d; // device array for the matrix A
    double *x, *e, *u, *v, *H, *temp;
    hipMallocManaged((void **) &A_d, m * n * sizeof(double));
    hipMallocManaged((void **) &x, m * sizeof(double));
    hipMallocManaged((void **) &e, m * sizeof(double));
    hipMallocManaged((void **) &u, m * sizeof(double));
    hipMallocManaged((void **) &v, m * sizeof(double));
    hipMallocManaged((void **) &H, m * m * sizeof(double));
    hipMallocManaged((void **) &temp, m * (n - 0) * sizeof(double));
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            A_d[i * n + j] = A_h[i][j];
        }
    }
    dim3 grid(1, 1);
    dim3 block(1, 1);
    householderQR_global<<<grid, block>>>(A_d, m, n, x, e, u, v, H, temp);
    hipDeviceSynchronize();
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            cout << A_d[i * n + j] << " ";
        }
        cout << endl;
    }
    hipFree(x);
    hipFree(e);
    hipFree(u);
    hipFree(v);
    hipFree(H);
    hipFree(temp);
    return 0;
}




